#include "hip/hip_runtime.h"
#include "cuda_irt.h"

#define   EPS     1.1929093e-7

/*

  constant variables declarations

*/


  __constant__    float         cutoff;
  __constant__    float         cutoff_sqrd;

  __constant__    grid_type     wg_type;
  __constant__    int           wg_x_bin;
  __constant__    int           wg_y_bin;
  __constant__    float         wg_x_rez;
  __constant__    float         wg_y_rez;
  __constant__    float         wg_x_amp;
  __constant__    float         wg_y_amp;


/*

  kernel declarations

*/

  __global__      void          compute_wigner_rd( float* quad, float* phase, float* wigner);
  __global__      void          compute_wigner_xy( float* quad, float* phase, float* wigner);
  __global__      void          compute_negativity( float* quad, float* neg); 

  __device__      float         eval_kernel(float);


EXT()  __host__ cuda_irt_margi_wk*     cuda_irt_margi_alloc
  (int n)
{
  
  cuda_irt_margi_wk* wk = (cuda_irt_margi_wk*) malloc( sizeof( cuda_irt_margi_wk ) ); 
  
  wk-> samples = n;

  size_t memory_size = n * sizeof(float2);

  hipHostMalloc( (void**) &wk-> host_data, memory_size)
  hipMalloc( (void**) &wk-> dev_data, memory_size );

  return wk;

}


EXT()  __host__ void                   cuda_irt_margi_free
  (cuda_irt_margi_wk* wk)
{

  hipHostFree( wk-> host_data );

  hipFree( wk-> dev_data );

  free( wk );

}


EXT()  __host__ void                   cuda_irt_margi_data
  (cuda_irt_margi_wk* wk, marginal* m)
{

  for( int i = 0; i < wk-> samples; i++)
    wk-> host_data[i] = make_float2( (float) m-> quad[i], (float) m-> phas[i] );

  size_t copy_size = wk-> samples * sizeof(float2);

  hipMemcpy( wk-> dev_data, wk-> host_data, copy_size, hipMemcpyHostToDevice  );
  
}


EXT()  __host__  void  cuda_irt_margi_grid 
  (grid* g)
{

  float wg_x_rez = (float) g-> x-> resolution;
  float wg_y_rez = (float) g-> y-> resolution;
  float wg_x_amp = (float) g-> x-> amplitude;
  float wg_y_amp = (float) g-> y-> amplitude;  

  hipMemcpyToSymbol(HIP_SYMBOL( "wg_type"),  &g-> grid_type, sizeof(grid_type));
  hipMemcpyToSymbol(HIP_SYMBOL( "wg_x_bin"), &g-> x-> n_val, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL( "wg_y_bin"), &g-> y-> n_val, sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL( "wg_x_rez"), &wg_x_rez, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( "wg_y_rez"), &wg_y_rez, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( "wg_x_amp"), &wg_x_amp, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( "wg_y_amp"), &wg_y_amp, sizeof(float));

}


EXT()  __host__  void  cuda_irt_margi_cutoff
  (Real cutoff)
{

  float cut = cutoff
  float cut_sqrd = (float) cutoff*cutoff;
  
  hipMemcpyToSymbol(HIP_SYMBOL( "cutoff"), &cut, sizeof(float));
  hipMemcpyToSymbol(HIP_SYMBOL( "cutoff_sqrd"), &cut_sqrd , sizeof(float));

}


EXT()  __host__  wigner*                 cuda_irt_margil
  (cuda_irt_margi_wk* wk)
{

  grid* g   = 
  wigner* w = wigner_alloc


}


EXT()  __host__  Real                    cuda_irt_margi_neg
  (cuda_irt_margi_wk* wk)
{


}


// Part 2 of 2: implement the fast kernel using shared memory
__global__ void reverseArrayBlock(int *d_out, int *d_in)
{

    extern __shared__ int s_data[];
    //__shared__ int s_data[256];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = ( gridDim.x - 1 - blockIdx.x )* blockDim.x + threadIdx.x;

    // Load one element per thread from device memory and store it 
    // *in reversed order* into temporary shared memory
    s_data[blockDim.x - 1 - threadIdx.x] = d_in[i]; 

    // Block until all threads in the block have written their data to shared mem
    __syncthreads();

    // write the data from shared memory in forward order, 
    // but to the reversed block offset as before
    d_out[j] = s_data[threadIdx.x]; 

}


__global__  void                         computer_wigner_rd
  ( float* quad, float* phase, float* wigner)
{

  extern __shared__ float quad[]; 
  extern __shared__ float phas[];

  float r = ;   //threadIdx
  float phi = ; //blockIdx
  float x;

  float w;

  for(int i = 0; i < nantoka; i++) {

    //copy data 

    __syncthreads();

    for (int n = 0; j < loaded memory size; j++) {

      x  = r * cosf( phi - phas[n] ) - quad[n];       
      w += eval_kernel(x);

    }


  }


}

__device__  float                        eval_kernel
  (float x)
{

  float p = (fabsf(x) + EPS)* cutoff;    
  float pp = 1.0f / p * p;

#ifdef FAST_KERNEL

  return pp * ( cutoff_sqrd * ( cosf(p) + p * sinf(p) ) - 1.0f );

#else

  return pp * ( cutoff_sqrd * ( __cosf(p) + p * __sinf(p) ) - 1.0f );

#endif

}

















int main( int argc, char** argv) 
{
    // pointer for host memory and size
    int *h_a;
    int dimA = 256 * 1024; // 256K elements (1MB total)

    // pointer for device memory
    int *d_b, *d_a;

    // define grid and block size
    int numThreadsPerBlock = 256;

    // Compute number of blocks needed based on array size and desired block size
    int numBlocks = dimA / numThreadsPerBlock;  

    // Part 1 of 2: Compute the number of bytes of shared memory needed
    // This is used in the kernel invocation below
    int sharedMemSize = numThreadsPerBlock * sizeof(int);

    // allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );

    // Initialize input array on host
    for (int i = 0; i < dimA; ++i)
    {
        h_a[i] = i;
    }

    // Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );

    // launch kernel
    dim3 dimGrid(numBlocks);
    dim3 dimBlock(numThreadsPerBlock);
    reverseArrayBlock<<< dimGrid, dimBlock, sharedMemSize >>>( d_b, d_a );
    //reverseArrayBlock<<< dimGrid, dimBlock >>>( d_b, d_a );

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");

    // device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );

    // Check for any CUDA errors
    checkCUDAError("memcpy");

    // verify the data returned to the host is correct
    for (int i = 0; i < dimA; i++)
    {
        assert(h_a[i] == dimA - 1 - i );
    }

    // free device memory
    hipFree(d_a);
    hipFree(d_b);

    // free host memory
    free(h_a);

    // If the program makes it this far, then the results are correct and
    // there are no run-time errors.  Good work!
    printf("Correct!\n");

    return 0;
}

#undef EPS
